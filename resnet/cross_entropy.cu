#include "hip/hip_runtime.h"
#include "cross_entropy.h"
#define PAR(total, threads) <<<((total) + threads - 1) / threads, threads>>>

__global__ void nll_loss(float *loss, const float *logits_grad, const int *labels, int C,
                         int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < N) {
        int class_id = labels[index];
        loss[index] =  -logits_grad[index * C + class_id];
    }
}

__global__ void nll_loss_backward(float *logits_grad, const float* loss_grad, const int *labels,
                                  int C, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < N * C) {
        logits_grad[index] = 0.0;
    }
    if(index < N) {
        int class_id = labels[index];
        logits_grad[index * C + class_id] = -0.001 * loss_grad[index];
    }
}

void CrossEntropy::forward(float *loss, const float *act, const int *labels) {
    auto kAlgo = HIPDNN_SOFTMAX_LOG;
    auto kMode = HIPDNN_SOFTMAX_MODE_INSTANCE;
    float one = 1.0, zero = 0.0;
    auto logits = (float *)global.get_workspace();
    hipdnnSoftmaxForward(global.cudnn_handle(), kAlgo, kMode, &one, dsc_io, act, &zero,
                        dsc_io, logits);
    nll_loss PAR(batch_size, 128)(loss, logits, labels, class_size, batch_size);
    // nll_loss
}

void CrossEntropy::backward(float *act_grad, const float* loss_grad, const int *labels) {
    auto kAlgo = HIPDNN_SOFTMAX_LOG;
    auto kMode = HIPDNN_SOFTMAX_MODE_INSTANCE;
    float one = 1.0, zero = 0.0;
    auto logits = (float *)global.get_workspace();
    // nll_loss
    nll_loss_backward PAR(class_size * batch_size, 128)(logits, loss_grad, labels,
                                                        class_size, batch_size);
    hipdnnSoftmaxBackward(global.cudnn_handle(), kAlgo, kMode, &one, dsc_io,
                         global.get_workspace(), dsc_io, logits, &zero, dsc_io, act_grad);
}
