#include "../doglib/graph/procedure.h"
#include "cg.h"
#include "components/cross_entropy.h"
#include <random>
#include <arpa/inet.h>
#include <thrust/count.h>
#include <fstream>
#include <thrust/device_vector.h>
struct functor {
    __host__ __device__ bool operator()(float x) {
        return x < 0.6931471805599453;
    }
};

// void dog_print(std::string name, const float* ptr, const dim_t& dim) {
//     cout << name << endl;
//     auto sz = get_volume(dim);

//     hipDeviceSynchronize();
//     host_vector<T> vec(sz);
//     hipMemcpy(vec.data(), ptr, sz * sizeof(float), hipMemcpyDefault);
//     auto tmp = dim;
//     std::reverse(tmp.begin(), tmp.end());
//     for(auto index : Range(sz)) {
//         int index_cpy = index;
//         for(auto x : tmp) {
//             if(index_cpy % x != 0) break;
//             index_cpy /= x;
//             cout << "--------" << endl;
//         }
//         cout << vec[index] << " ";
//     }
//     cout << endl << "##########" << endl;
// }

void dog_log(float* ptr, const dim_t& dim) {}
using std::vector;
const char* data_file = "/home/guilin/workspace/data/mnist/images-idx3-ubyte";
const char* labels_file = "/home/guilin/workspace/data/mnist/labels-idx3-ubyte";

host_vector<float> get_data() {
    host_vector<float> data;
    std::ifstream fin(data_file);
    int magic, number, w, h;
    fin >> magic >> number >> w >> h;
    number = htonl(number);
    h = htonl(h);
    w = htonl(w);
    assert(htonl(magic) == 0x00000801);
    assert(number == 60000);
    assert(h == 28);
    assert(w == 28);
    number = 600;
    data.resize(number);
    for(auto id : Range(number)) {
        uint8_t x;
        fin >> x;
        data[id] = x / 128.0 - 0.5;
    }
    return data;
}

host_vector<float> get_labels() {
    host_vector<float> data;
    std::ifstream fin(labels_file);
    int magic, number;
    fin >> magic >> number;
    number = htonl(number);
    assert(htonl(magic) == 0x00000803);
    assert(number == 60000);
    number = 600;
    data.resize(number);
    for(auto id : Range(number)) {
        uint8_t x;
        fin >> x;
        data[id] = x;
    }
    return data;
}

Global global;
int main() {
    Engine eng;
    // define network structure
    int B = 600;
    int features = 28 * 28;
    int hidden = features;
    int classes = 10;
    dim_t input_dim = {B, features};

    auto x = eng.insert_leaf<PlaceHolderNode>(input_dim);
    eng.src_node = x;
    auto shortcut = x;
    x = eng.insert_node<FCNode>(x, B, features, hidden);
    x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    x = eng.insert_node<FCNode>(x, B, hidden, hidden);
    x = eng.insert_blend<AddNode>(x, shortcut, dim_t{B, hidden});
    x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    x = eng.insert_node<FCNode>(x, B, hidden, classes);
    eng.dest_node = x;
    eng.finish_off();

    host_vector<float> input = get_data();
    // input.resize(B * 1000);
    // std::default_random_engine e(201);
    // for(auto& x : input) {
    //     x = (float)(e() % 10001) / 5000 - 1;
    // }

    host_vector<int> labels = get_labels();
    // for(auto id : Range(B)) {
    //     float sum = 0;
    //     for(auto x : Range(features)) {
    //         sum *= input[id * features + x];
    //     }
    //     int label = sum >= 0 ? 1 : 0;
    //     labels.push_back(label);
    // }

    // for(auto x : labels) {
    //     cout << x << " ";
    // }
    cout << endl;

    device_vector<int> dev_labels = labels;
    DeviceVector<T> losses(B);
    CrossEntropy ce(B, classes);
    global.update_workspace_size(ce.workspace());
    for(auto x : Range(10000)) {
        eng.zero_grad();
        eng.forward_pass(input.data());
        auto act = eng.get_ptr(eng.dest_node);
        auto act_grad = eng.get_ptr(~eng.dest_node);

        ce.forward(losses, act, dev_labels.data().get());
        // dog_print("##", act, dim_t{B, classes});
        auto loss = thrust::reduce(thrust::device, losses.begin(), losses.end());
        ce.backward(act_grad, 0.5, losses, dev_labels.data().get());
        // dog_print("SS", act_grad, dim_t{B, classes});
        // // dog_print("hhd", act, {B});

        eng.backward_pass(act_grad);
        int correct = thrust::count_if(losses.begin(), losses.end(), functor());
        eng.step();
        cout << loss / B << " " << correct << endl;
    }
}