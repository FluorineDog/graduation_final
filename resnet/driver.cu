#include "../doglib/graph/procedure.h"
#include "cg.h"
#include "components/cross_entropy.h"
#include <random>

#include <thrust/count.h>
#include <thrust/device_vector.h>
struct functor {
    __host__ __device__ bool operator()(float x) {
        return x < 0.6931471805599453;
    }
};

// void dog_print(std::string name, const float* ptr, const dim_t& dim) {
//     cout << name << endl;
//     auto sz = get_volume(dim);

//     hipDeviceSynchronize();
//     host_vector<T> vec(sz);
//     hipMemcpy(vec.data(), ptr, sz * sizeof(float), hipMemcpyDefault);
//     auto tmp = dim;
//     std::reverse(tmp.begin(), tmp.end());
//     for(auto index : Range(sz)) {
//         int index_cpy = index;
//         for(auto x : tmp) {
//             if(index_cpy % x != 0) break;
//             index_cpy /= x;
//             cout << "--------" << endl;
//         }
//         cout << vec[index] << " ";
//     }
//     cout << endl << "##########" << endl;
// }

void dog_log(float* ptr, const dim_t& dim){

}

Global global;
int main() {
    Engine eng;
    // define network structure
    int B = 10000;
    int features = 8;
    int hidden = features;
    int classes = 2;
    dim_t input_dim = {B, features};

    auto x = eng.insert_leaf<PlaceHolderNode>(input_dim);
    eng.src_node = x;
    auto shortcut = x;
    x = eng.insert_node<FCNode>(x, B, features, hidden);
    x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    x = eng.insert_node<FCNode>(x, B, hidden, hidden);
    x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    x = eng.insert_blend<AddNode>(x, shortcut, dim_t{B, hidden});
    x = eng.insert_node<FCNode>(x, B, hidden, classes);
    eng.dest_node = x;
    eng.finish_off();

    host_vector<float> input;
    input.resize(B * 1000);
    std::default_random_engine e(201);
    for(auto& x : input) {
        x = (float)(e() % 10001) / 5000 - 1;
    }

    host_vector<int> labels;
    for(auto id : Range(B)) {
        float sum = 0;
        for(auto x : Range(features)) {
            sum += input[id * features + x];
        }
        int label = sum >= 0 ? 1 : 0;
        labels.push_back(label);
    }

    for(auto x : labels) {
        cout << x << " ";
    }
    cout << endl;

    device_vector<int> dev_labels = labels;
    DeviceVector<T> losses(B);
    CrossEntropy ce(B, classes);
    global.update_workspace_size(ce.workspace());
    for(auto x : Range(100)) {
        eng.zero_grad();
        eng.forward_pass(input.data());
        auto act = eng.get_ptr(eng.dest_node);
        auto act_grad = eng.get_ptr(~eng.dest_node);

        ce.forward(losses, act, dev_labels.data().get());
        // dog_print("##", act, dim_t{B, classes});
        auto loss = thrust::reduce(thrust::device, losses.begin(), losses.end());
        ce.backward(act_grad, 0.0001, losses, dev_labels.data().get());
        // dog_print("SS", act_grad, dim_t{B, classes});
        // // dog_print("hhd", act, {B});
        
        eng.backward_pass(act_grad);
        int correct = thrust::count_if(losses.begin(), losses.end(), functor());
        eng.step();
        cout << "^^" << loss / B << "%%" << correct << endl;
    }
}