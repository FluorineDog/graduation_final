#include "../doglib/graph/procedure.h"
#include "cg.h"
#include "components/cross_entropy.h"
#include <random>
#include <arpa/inet.h>
#include <thrust/count.h>
#include <thrust/reduce.h>
#include <fstream>
#include <thrust/device_vector.h>
struct functor {
    __host__ __device__ bool operator()(float x) {
        return x < 0.6931471805599453;
    }
};

// void dog_print(std::string name, const float* ptr, const dim_t& dim) {
//     cout << name << endl;
//     auto sz = get_volume(dim);

//     hipDeviceSynchronize();
//     host_vector<T> vec(sz);
//     hipMemcpy(vec.data(), ptr, sz * sizeof(float), hipMemcpyDefault);
//     auto tmp = dim;
//     std::reverse(tmp.begin(), tmp.end());
//     for(auto index : Range(sz)) {
//         int index_cpy = index;
//         for(auto x : tmp) {
//             if(index_cpy % x != 0) break;
//             index_cpy /= x;
//             cout << "--------" << endl;
//         }
//         cout << vec[index] << " ";
//     }
//     cout << endl << "##########" << endl;
// }

void dog_log(float* ptr, const dim_t& dim) {}
using std::vector;
const char* data_file = "/home/guilin/workspace/data/mnist/images-idx3-ubyte";
const char* labels_file = "/home/guilin/workspace/data/mnist/labels-idx1-ubyte";

host_vector<float> get_data() {
    host_vector<float> data;
    std::ifstream fin(data_file, std::ios::binary);
    uint32_t magic, number, w, h;
    fin.read((char*)&magic, 4);
    fin.read((char*)&number, 4);
    fin.read((char*)&w, 4);
    fin.read((char*)&h, 4);
    magic = htonl(magic);
    number = htonl(number);
    h = htonl(h);
    w = htonl(w);
    assert(magic == 0x00000803);
    assert(number == 60000);
    assert(h == 28);
    assert(w == 28);
    number = 60000;
    auto sz = number * w * h;
    data.resize(sz);
    vector<uint8_t> buffer(sz);
    fin.read((char*)buffer.data(), sz);
    for(auto id : Range(sz)) {
        uint8_t x = buffer[id];
        assert(0 <= x && x < 256);
        data[id] = x / 255.0;
    }
    return data;
}

host_vector<int> get_labels() {
    host_vector<int> data;
    std::ifstream fin(labels_file, std::ios::binary);
    uint32_t magic, number;
    fin.read((char*)&magic, 4);
    fin.read((char*)&number, 4);
    magic = htonl(magic);
    number = htonl(number);
    assert(magic == 0x00000801);
    assert(number == 60000);
    number = 60000;
    auto sz = number;
    data.resize(sz);
    vector<uint8_t> buffer(sz);
    fin.read((char*)buffer.data(), sz);
    for(auto id : Range(sz)) {
        uint8_t x = buffer[id];
        assert(0 <= x && x < 10);
        data[id] = x;
    }
    return data;
}

float get_acc(float* dev_logits, int* labels, int N, int feature) {
    vector<float> buffer(N * feature);
    hipMemcpy(buffer.data(), dev_logits, N * feature * sizeof(float), hipMemcpyDefault);
    int count = 0;
    for(int b : Range(N)) {
        auto loc = std::max_element(buffer.begin() + b * feature,
                                    buffer.begin() + (b + 1) * feature) -
                   buffer.begin() - b * feature;
        //
        assert(0 <= loc && loc < feature);
        assert(0 <= loc && loc < feature);
        count += (loc == labels[b]) ? 1 : 0;
    }
    return count * 1.0 / N;
}

Global global;
int main() {
    Engine eng;
    // define network structure
    int B = 1;
    int features = 4;
    // int hidden = 28 * 28;
    int hidden = 4;
    int classes = 2;
    dim_t input_dim = {B, features};

    auto x = eng.insert_leaf<PlaceHolderNode>(input_dim);
    eng.src_node = x;

    // auto shortcut = x;
    // x = eng.insert_node<FCNode>(x, B, features, hidden);
    // x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    // x = eng.insert_node<FCNode>(x, B, hidden, hidden);
    // x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    // x = eng.insert_node<FCNode>(x, B, hidden, hidden);
    // x = eng.insert_node<ActivationNode>(x, dim_t{B, hidden});
    // x = eng.insert_blend<AddNode>(x, shortcut, dim_t{B, hidden});

    x = eng.insert_node<FCNode>(x, B, hidden, classes);
    eng.dest_node = x;
    eng.finish_off();
    
    // auto total = 60000;
    // host_vector<float> data_raw = get_data();
    // host_vector<int> labels_raw = get_labels();

    auto total = B;
    host_vector<float> data_raw;
    host_vector<int> labels_raw;
    data_raw.resize(B * 1000);
    std::default_random_engine e(201);
    for(auto& x : data_raw) {
        x = (float)(e() % 10001) / 5000 - 1;
    }
    for(auto id : Range(B)) {
        float sum = 0;
        for(auto x : Range(features)) {
            sum += data_raw[id * features + x];
        }
        int label = sum >= 0 ? 1 : 0;
        labels_raw.push_back(label);
    }

    for(auto x: labels_raw){
        cout << x; 
    }  
    cout << endl;

    DeviceVector<T> losses(B);
    CrossEntropy ce(B, classes);
    global.update_workspace_size(ce.workspace());
    for(auto x : Range(3)) {
        auto offset_lb = x % (total / B) * B;
        auto offset_dt = offset_lb * features;
        auto data_beg = data_raw.data() + offset_dt;
        auto data_end = data_raw.data() + offset_dt + B * features;
        auto labels_beg = labels_raw.data() + offset_lb;
        auto labels_end = labels_raw.data() + offset_lb + B;
        eng.zero_grad();
        eng.forward_pass(data_beg);
        auto act = eng.get_ptr(eng.dest_node);
        auto act_grad = eng.get_ptr(~eng.dest_node);
        device_vector<int> dev_labels(labels_beg, labels_end);
        dog_print("##", act, dim_t{B, classes});
        ce.forward(losses, act, dev_labels.data().get());
        // eng.get_mm().l2_forward(losses, B, 0.1);
        // dog_print("??", losses, dim_t{B});
        auto loss = thrust::reduce(thrust::device, losses.begin(), losses.end());

        // eng.get_mm().l2_backward(losses, B, 0.1);
        ce.backward(act_grad, 0.1, act, losses, dev_labels.data().get());
        // dog_print("SS", act_grad, dim_t{B, classes});
        // dog_print("hhd", act, {B});

        eng.backward_pass(act_grad);
        // auto correct = thrust::count_if(losses.begin(), losses.end(), functor());
        auto correct = get_acc(act, labels_beg, B, classes);
        if(loss != loss) {
            break;
        }
        if(x % 100) {
            eng.step();
            cout << loss / B << " " << correct << endl;
        } else {
            cout << "test: " << loss / B << " " << correct << endl;
        }
    }
}